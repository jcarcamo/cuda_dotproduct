/*
 * blockAndThread.cu
 * includes setup funtion called from "driver" program
 * also includes kernel function 'cu_fillArray()'
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 1024

__global__ void kernel_dotproduct(long long *force_d, long long *distance_d, long long *result_d, long long size) {
    extern __shared__ long long sadata[];
    
    int n = blockDim.x;
    int nTotalThreads;
    if (!n){
	nTotalThreads = n;
    }else{
	//(0 == 2^0)
    	int x = 1;
    	while(x < n)
    	{
      	    x <<= 1;
    	}
        nTotalThreads = x;
    }

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    long long i = blockIdx.x*nTotalThreads + threadIdx.x;
    if(i < size){
    	sadata[tid] = force_d[i]*distance_d[i];
    }
    __syncthreads();
    
    // do reduction in shared mem
    //if(i < size){
    for (unsigned int s=1; s < nTotalThreads; s *= 2) {
        if (tid % (2*s) == 0 && (tid+s) < size) {
            sadata[tid] += sadata[tid + s];
        }
        __syncthreads();
    }
    //}  
    // write result for this block to global mem
    if (tid == 0) result_d[blockIdx.x] = sadata[0];
}

template <unsigned int blockSize>
__global__ void kernel_dotproduct2(long long *force_d, long long *distance_d, long long *result_d, long long size)
{
	extern __shared__ long long sdata[];
	int n = blockDim.x;
    	int nTotalThreads;
    	if (!n){
        	nTotalThreads = n;
    	}else{
        	//(0 == 2^0)
        	int x = 1;
        	while(x < n)
        	{
            		x <<= 1;
        	}
        	nTotalThreads = x;
    	}

	unsigned int tid = threadIdx.x;
	long long i = blockIdx.x*(nTotalThreads*2) + threadIdx.x;
	if((i+nTotalThreads)< size){
		sdata[tid] = force_d[i]*distance_d[i] + force_d[i+nTotalThreads]*distance_d[i+nTotalThreads] ;
	} else {
		if(i < size){
			sdata[tid] = force_d[i]*distance_d[i];
		}else{
			sdata[tid] = 0;
		}
	}
	__syncthreads();
	for (long long s=nTotalThreads/2; s>32 && (tid+s) < size; s>>=1)
	{
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}

	// write result for this block to global mem
	if (tid == 0) result_d[blockIdx.x] = sdata[0];
	
}

// The __global__ directive identifies this function as a kernel
// Note: all kernels must be declared with return type void 
__global__ void kernel_check_threads (long long *force_d, long long *distance_d)
{
    long long x;

    // Note: CUDA contains several built-in variables
    // blockIdx.x returns the blockId in the x dimension
    // threadIdx.x returns the threadId in the x dimension
    x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    force_d[x] = blockIdx.x;
    distance_d[x] = threadIdx.x;
}


// This function is called from the host computer.
// It manages memory and calls the function that is executed on the GPU
extern "C" void cuda_dotproduct (long long *force, long long *distance, long long arraySize, long long *result_array, long long *result)
{
	// block_d and thread_d are the GPU counterparts of the arrays that exists in host memory 
	long long *force_d;
	long long *distance_d;
	long long *result_d;

	hipError_t op_result;
	
	// allocate space in the device 
	op_result = hipMalloc ((void**) &force_d, sizeof(long long) * arraySize);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMalloc (foce) failed.");
		exit(1);
	}
	op_result = hipMalloc ((void**) &distance_d, sizeof(long long) * arraySize);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMalloc (distance) failed.");
		exit(1);
	}
	op_result = hipMalloc ((void**) &result_d, sizeof(long long)*arraySize);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipMalloc (result) failed.");
                exit(1);
        }
	
	//copy the arrays from host to the device 
	op_result = hipMemcpy (force_d, force, sizeof(long long) * arraySize, hipMemcpyHostToDevice);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (force) failed.");
		exit(1);
	}
	op_result = hipMemcpy (distance_d, distance, sizeof(long long) * arraySize, hipMemcpyHostToDevice);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (distance) failed.");
		exit(1);
	}
        
	op_result = hipMemcpy (result_d, result_array, sizeof(long long) * arraySize, hipMemcpyHostToDevice);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipMemcpy host->dev (result) failed.");
                exit(1);
        }
	
	int threads;
	if (arraySize < 256 ){
		threads = 128;
	} else if (arraySize < 512){
		threads = 256;
	} else if (arraySize < 1024){
		threads = 512;
	} else {
		threads = BLOCK_SIZE;
	}
	long long block_size = threads;
        long long blocks = ceil(arraySize / ((float) block_size));
	// set execution configuration
        dim3 dimblock (block_size);
        dim3 dimgrid (blocks);
        int smemSize = dimblock.x * sizeof(long long);
        // actual computation: Call the kernel
	//kernel_dotproduct<<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
        switch (threads)
	{
		case 128:
		  kernel_dotproduct2<128><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
		  break;
		case 256:
                  kernel_dotproduct2<256><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
                  break;
		case 512:
                  kernel_dotproduct2<256><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize);
                  break;
		default:
		 kernel_dotproduct2<BLOCK_SIZE><<<dimgrid,dimblock,smemSize>>>(force_d, distance_d, result_d, arraySize); 
		 break;
	}
	//kernel_check_threads<<<dimgrid,dimblock>>>(force_d, distance_d);
        // transfer results back to host
	op_result = hipMemcpy (force, force_d, sizeof(long long) * arraySize, hipMemcpyDeviceToHost);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (force) failed.");
		exit(1);
	}
	op_result = hipMemcpy (distance, distance_d, sizeof(long long) * arraySize, hipMemcpyDeviceToHost);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host <- dev (distance) failed.");
		exit(1);
	}

	op_result = hipMemcpy (result_array, result_d, sizeof(long long)*arraySize, hipMemcpyDeviceToHost);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipMemcpy host <- dev (result) failed.");
                exit(1);
        }
	
	int i,j = 0;
	for (i = 0; i < arraySize; i++){
		if(result_array[i] < 0){
			j++;
		}
	}
	printf("faulty # = %d \n",j);

	// release the memory on the GPU 
	op_result = hipFree (force_d);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipFree (force) failed.");
		exit(1);
	}
	op_result = hipFree (distance_d);
	if (op_result != hipSuccess) {
		fprintf(stderr, "hipFree (distance) failed.");
		exit(1);
	}
	op_result = hipFree (result_d);
        if (op_result != hipSuccess) {
                fprintf(stderr, "hipFree (distance) failed.");
                exit(1);
        }
        
}

